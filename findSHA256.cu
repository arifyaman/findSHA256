#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <pthread.h>
#include <locale.h>
#include "lib/sha256.cuh"

#define THREADS 1200
#define BLOCKS 256

#define MAX_HASH_RESULTS 40
#define MAX_USER_INPUT MAX_HASH_RESULTS *SHA256_BLOCK_SIZE * 2 + (MAX_HASH_RESULTS - 1)

#define BYTE_LENGTH_TO_BE_FOUND 8

__global__ void sha256_cuda(BYTE *solution, int *blockContainsSolution, unsigned long baseSeed, int userHashCount, WORD *d_realHashedState, BYTE *allowedBytes)
{
    SHA256_CTX ctx;
    baseSeed ^= (unsigned long)blockIdx.x * blockDim.x + threadIdx.x;

    sha256_init(&ctx);
    BYTE i = 0;

    for (i = 0; i < BYTE_LENGTH_TO_BE_FOUND; ++i)
    {
        baseSeed = ((baseSeed << 13) ^ baseSeed ^ (baseSeed >> 17) ^ (baseSeed << 5));
        sha256_update(&ctx, &allowedBytes[baseSeed & 0x0F], 1);
    }
    
    sha256_final(&ctx);

    for (BYTE j = 0; j < userHashCount; ++j)
    {
        for (i = 0; i < 8; ++i)
        {
            if (d_realHashedState[8 * j + i] != ctx.state[i])
            {
                break;
            }
            if (i == 7)
            {
                if (*blockContainsSolution == 1)
                    return;

                *blockContainsSolution = 1;
                for (i = 0; i < BYTE_LENGTH_TO_BE_FOUND; ++i)
                    solution[i] = ctx.data[i];

                return;
            }
        }
    }
}

void reverseBytes(BYTE *bytes, int length)
{
    int i;
    for (i = 0; i < length / 2; ++i)
    {
        BYTE temp = bytes[i];
        bytes[i] = bytes[length - i - 1];
        bytes[length - i - 1] = temp;
    }
}

void retrieveState(const BYTE *hash, WORD *state)
{
    int i, j;
    BYTE temp[4];

    for (i = 0, j = 0; i < 32; i += 4, ++j)
    {
        // Copy 4 bytes from the hash into temp array
        temp[0] = hash[i];
        temp[1] = hash[i + 1];
        temp[2] = hash[i + 2];
        temp[3] = hash[i + 3];

        // Reverse the byte order
        reverseBytes(temp, 4);

        // Combine the bytes to form a uint32_t integer
        state[j] = *(WORD *)temp;
    }
}

bool hexStringToShaStateWords(const char *hexString, WORD *words)
{
    BYTE bytes[SHA256_BLOCK_SIZE];

    for (int i = 0; i < 32; ++i)
    {
        bytes[i] = (BYTE)(((hexString[i * 2] & 0xF) + ((hexString[i * 2] >> 6) * 9)) << 4);
        bytes[i] |= (BYTE)((hexString[i * 2 + 1] & 0xF) + ((hexString[i * 2 + 1] >> 6) * 9));
    }

    retrieveState(bytes, words);

    return true;
}

long long timeInMilliseconds(void)
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

int main()
{
    hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice);

    unsigned char *d_solution;
    hipMalloc((void **)&d_solution, sizeof(char) * BYTE_LENGTH_TO_BE_FOUND);

    int *blockContainsSolution = (int *)malloc(sizeof(int));
    int *d_blockContainsSolution;
    hipMalloc(&d_blockContainsSolution, sizeof(int));

    BYTE allowedBytes[16] = {0x30, 0x31, 0x32, 0x33, 0x34, 0x35, 0x36, 0x37, 0x38, 0x39, 0x61, 0x62, 0x63, 0x64, 0x65, 0x66};
    unsigned char *d_allowedBytes;
    hipMalloc((void **)&d_allowedBytes, sizeof(BYTE) * 16);
    hipMemcpy(d_allowedBytes, allowedBytes, sizeof(BYTE) * 16, hipMemcpyHostToDevice);

    // Get the expected hash result convert to WORD in big endian byte ordering to compare with sha256 context state later and allocate it on the GPU
    char userHashResults[MAX_USER_INPUT];
    char *tokens[MAX_USER_INPUT];
    int userHashCount = 0;
    scanf("%[^\n]%*c", userHashResults);
    char *token = strtok(userHashResults, ",");
    while (token != NULL && userHashCount < MAX_HASH_RESULTS)
    {
        tokens[userHashCount++] = token;
        token = strtok(NULL, ",");
    }
    WORD *d_realHashedState;
    hipMalloc((void **)&d_realHashedState, sizeof(WORD) * 8 * userHashCount);

    WORD wordResult[8];
    for (int i = 0; i < userHashCount; ++i)
    {
        hexStringToShaStateWords(tokens[i], wordResult);
        hipMemcpy(d_realHashedState + 8 * i, wordResult, sizeof(WORD) * 8, hipMemcpyHostToDevice);
    }

    unsigned long hashCount = 0;
    long long start = timeInMilliseconds();
    long long seed = start;

    while (1)
    {
        seed = ((seed << 13) ^ seed ^ (seed >> 17) ^ (seed << 5));
        hashCount += THREADS * BLOCKS;
        sha256_cuda<<<THREADS, BLOCKS>>>(d_solution, d_blockContainsSolution, seed, userHashCount, d_realHashedState, d_allowedBytes);

        // hipDeviceSynchronize();

        hipMemcpy(blockContainsSolution, d_blockContainsSolution, sizeof(int), hipMemcpyDeviceToHost);
        if (*blockContainsSolution == 1)
        {
            BYTE solution[BYTE_LENGTH_TO_BE_FOUND];
            hipMemcpy(solution, d_solution, sizeof(BYTE) * BYTE_LENGTH_TO_BE_FOUND, hipMemcpyDeviceToHost);

            long elapsed = timeInMilliseconds() - start;
            printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\n", hashCount, ((float)elapsed) / 1000.0, (unsigned long)((double)hashCount / (double)elapsed) * 1000);
            printf("Hash Found\n");
            for (int i = 0; i < BYTE_LENGTH_TO_BE_FOUND; ++i)
            {
                printf("%c", (char)solution[i]);
            }
            printf("\n");
            break;
        }
        long elapsed = timeInMilliseconds() - start;
        printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\r", hashCount, ((float)elapsed) / 1000.0, (unsigned long)((double)hashCount / (double)elapsed) * 1000);
    }
}